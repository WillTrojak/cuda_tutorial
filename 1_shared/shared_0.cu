#include "hip/hip_runtime.h"
#include <algorithm>
#include <stdio.h>

#include <hip/hip_runtime.h>

typedef float ftype;

void error_check(hipError_t cerr, int id) {
    if (cerr != hipSuccess) {
        fprintf(stderr, "Failed (%d) with error code %s\n", id, hipGetErrorString(cerr));
        exit(EXIT_FAILURE);
    }
}


// z[n * m] = sigma(w[n * k] * x[k * m] + b[n])
// -> add the linear layer with bias and ReLU activation. I've already given all
//    the for loops you'll need.
//    Note: std::max
void layer_test(int n, int m, int k, const ftype *w, const ftype *x,
                const ftype *b, ftype *z) {
    for (int i=0; i<m; ++i) {
        for (int ii=0; ii<n; ++ii) {
            for (int iii=0; iii<k; ++iii) {

            }
        }
    }
}


// z = sigma(w*x + b)
__global__ void layer_32x32(const int n, const ftype __restrict__ *w,
                            const ftype __restrict__ *x,
                            const ftype __restrict__ *b,
                            ftype __restrict__ *z) {
    // -> Calculate your global thread index, j
    // -> Calcualte the batch and row.

    if (j < n) {
        // -> Initialise an accumulator with the bias

        // -> get each thread to perform one of the dot-producs

        // -> ReLU
        // -> Set z
    }
}


int main(void) {
    hipError_t cerr;

    // Some constants and sizes
    const int nbatch = 1024;
    const int nfeat = 32;
    const int n = nbatch * nfeat;
    size_t size_w = nfeat * nfeat * sizeof(ftype);
    size_t size_b = nfeat * sizeof(ftype);
    size_t size_x = n * sizeof(ftype);
    size_t size_z = n * sizeof(ftype);

    // Allocate the host memory
    ftype *w_h, *b_h, *x_h, *z_h, *z_t;

    w_h = (ftype *)malloc(size_w);
    b_h = (ftype *)malloc(size_b);
    x_h = (ftype *)malloc(size_x);
    z_h = (ftype *)malloc(size_z);
    z_t = (ftype *)malloc(size_z);

    // Initialise the weights and biases on the host
    for (int i = 0; i < nfeat; ++i) {
        b_h[i] = 2*(rand() / (ftype)RAND_MAX) - 1;
        for (int ii = 0; ii < nfeat; ++ii)
            w_h[ii + nfeat*i] = 2*(rand() / (ftype)RAND_MAX) - 1;
    }

    // Initialise the data
    for (int i = 0; i < nbatch; ++i)
        for (int ii = 0; ii < nfeat; ++ii)
            x_h[ii + nfeat*i] = 2*(rand() / (ftype)RAND_MAX) - 1;

    // Allocate the device vectors
    ftype *w_d, *b_d, *x_d, *z_d;
    cerr = hipMalloc((void **)&w_d, size_w);
    error_check(cerr, 0);

    cerr = hipMalloc((void **)&b_d, size_b);
    error_check(cerr, 1);

    cerr = hipMalloc((void **)&x_d, size_x);
    error_check(cerr, 2);

    cerr = hipMalloc((void **)&z_d, size_z);
    error_check(cerr, 3);

    // Copy the weights, biases, and data to the host
    cerr = hipMemcpy(w_d, w_h, size_w, hipMemcpyHostToDevice);
    error_check(cerr, 4);
    cerr = hipMemcpy(b_d, b_h, size_b, hipMemcpyHostToDevice);
    error_check(cerr, 5);
    cerr = hipMemcpy(x_d, x_h, size_x, hipMemcpyHostToDevice);
    error_check(cerr, 6);

    // Calcualte the thread block and grid sizes
    int threads_per_block = 128;
    int blocks_per_grid = (n + threads_per_block - 1) / threads_per_block;
    printf("Threads per block: %d\nBlocks per grid: %d\n",
           threads_per_block,
           blocks_per_grid);

    // Launch the kernel
    // -> Add launch of DEVICE kernel

    // Copy result to host
    cerr = hipMemcpy(z_h, z_d, size_z, hipMemcpyDeviceToHost);
    error_check(cerr, 7);

    // Verify the result
    // -> Add launch of the HOST version so we can test
    for (int i=0; i < nbatch; ++i)
        for (int ii=0; ii < nfeat; ++ii) {
            if (fabs(z_h[ii + i*nfeat] - z_t[ii + i*nfeat]) > 1e-5) {
                printf("value: %f test: %f\n", z_h[i], z_t[i]);
                printf("Result incorrect in element: (%d %d)\n", ii, i);
                exit(-1);
            }
        }

    printf("Test passed\n");

    // Free device memory
    cerr = hipFree(w_d);
    error_check(cerr, 6);

    cerr = hipFree(b_d);
    error_check(cerr, 7);

    cerr = hipFree(x_d);
    error_check(cerr, 8);

    cerr = hipFree(z_d);
    error_check(cerr, 9);

    // Free host memory
    free(w_h);
    free(b_h);
    free(x_h);
    free(z_h);
    free(z_t);

    return 0;
}
