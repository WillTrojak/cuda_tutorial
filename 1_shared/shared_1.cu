#include "hip/hip_runtime.h"
#include <stdio.h>
#include <algorithm>

#include <hip/hip_runtime.h>

typedef float ftype;

void error_check(hipError_t cerr, int id) {
    if (cerr != hipSuccess) {
        fprintf(stderr, "Failed (%d) with error code %s\n", id, hipGetErrorString(cerr));
        exit(EXIT_FAILURE);
    }
}


// z[n * m] = sigma(w[n * k] * x[k * m] + b[n])
void layer_test(int n, int m, int k, const ftype *w, const ftype *x,
                const ftype *b, ftype *z) {
    for (int i=0; i<m; ++i)
        for (int ii=0; ii<n; ++ii){
            ftype a = b[ii];
            for (int iii=0; iii<k; ++iii)
                a += w[ii + iii*n]*x[iii + i*k];
            z[ii + i*n] = std::max((ftype)0., a);
        }
}


// z = sigma(w*x + b)
__global__ void layer_32x32_shared(const nt n, const ftype __restrict__ *w,
                                   const ftype __restrict__ *x,
                                   const ftype __restrict__ *b,
                                   ftype __restrict__ *z) {
    int j = blockDim.x * blockIdx.x + threadIdx.x;
    int batch = j / 32;
    // -> set the row number
    int row = threadIdx.x % warpSize;
    // -> Allocate the shared memory
    __shared__ ftype w_s[32*32], b_s[32];

    if (j < n) {
        // Load weight and bias into shared memory
        // -> load weight and bias into sahred memory

        // -> Do we need to do anything before moving onto the calcualtion?
        //    Are we sure them load is done?

        // Initialise an accumulator with the bias
        ftype acc = b_s[row];
        for (int i=0; i < 32; i++)
            // -> Change the dot product to use shared memory.

        // Apply ReLU activation and write to z
        z[row + 32*batch] = max((ftype)0., acc);
    }
}


int main(void) {
    hipError_t cerr;

    const int nbatch = 64;
    const int nfeat = 32;
    const int n = nbatch * nfeat;
    size_t size_w = nfeat * nfeat * sizeof(ftype);
    size_t size_b = nfeat * sizeof(ftype);
    size_t size_x = n * sizeof(ftype);
    size_t size_z = n * sizeof(ftype);

    // Allocate the host memory
    ftype *w_h, *b_h, *x_h, *z_h, *z_t;

    w_h = (ftype *)malloc(size_w);
    b_h = (ftype *)malloc(size_b);
    x_h = (ftype *)malloc(size_x);
    z_h = (ftype *)malloc(size_z);
    z_t = (ftype *)malloc(size_z);

    // Initialise the weights and biases on the host
    for (int i = 0; i < nfeat; ++i) {
        b_h[i] = 2*(rand() / (ftype)RAND_MAX) - 1;
        for (int ii = 0; ii < nfeat; ++ii)
            w_h[ii + nfeat*i] = 2*(rand() / (ftype)RAND_MAX) - 1;
    }

    // Initialise the data
    for (int i = 0; i < nbatch; ++i)
        for (int ii = 0; ii < nfeat; ++ii)
            x_h[ii + nfeat*i] = 2*(rand() / (ftype)RAND_MAX) - 1;


    // Allocate the device vectors
    ftype *w_d, *b_d, *x_d, *z_d;
    cerr = hipMalloc((void **)&w_d, size_w);
    error_check(cerr, 0);

    cerr = hipMalloc((void **)&b_d, size_b);
    error_check(cerr, 1);

    cerr = hipMalloc((void **)&x_d, size_x);
    error_check(cerr, 2);

    cerr = hipMalloc((void **)&z_d, size_z);
    error_check(cerr, 3);

    // Copy the weights, biases, and data to the host
    cerr = hipMemcpy(w_d, w_h, size_w, hipMemcpyHostToDevice);
    error_check(cerr, 4);
    cerr = hipMemcpy(b_d, b_h, size_b, hipMemcpyHostToDevice);
    error_check(cerr, 5);
    cerr = hipMemcpy(x_d, x_h, size_x, hipMemcpyHostToDevice);
    error_check(cerr, 6);

    // Calcualte the thread block and grid sizes
    int threads_per_block = 128;
    int blocks_per_grid = (n + threads_per_block - 1) / threads_per_block;
    printf("Threads per block: %d\nBlocks per grid: %d\n",
           threads_per_block,
           blocks_per_grid);

    // Launch the kernel
    layer_32x32_shared<<<blocks_per_grid, threads_per_block>>>(n, w_d, x_d, b_d, z_d);

    // Copy result to host
    cerr = hipMemcpy(z_h, z_d, size_z, hipMemcpyDeviceToHost);
    error_check(cerr, 7);

    // Verify the result
    layer_test(nfeat, nbatch, nfeat, w_h, x_h, b_h, z_t);
    for (int i=0; i < nbatch; ++i)
        for (int ii=0; ii < nfeat; ++ii) {
            if (fabs(z_h[ii + i*nfeat] - z_t[ii + i*nfeat]) > 1e-5) {
                printf("value: %f test: %f\n", z_h[i], z_t[i]);
                printf("Result incorrect in element: (%d %d)\n", ii, i);
                exit(-1);
            }
        }

    printf("Test passed\n");

    // Free device memory
    cerr = hipFree(w_d);
    error_check(cerr, 6);

    cerr = hipFree(b_d);
    error_check(cerr, 7);

    cerr = hipFree(x_d);
    error_check(cerr, 8);

    cerr = hipFree(z_d);
    error_check(cerr, 9);

    // Free host memory
    free(w_h);
    free(b_h);
    free(x_h);
    free(z_h);
    free(z_t);

    return 0;
}
