#include "hip/hip_runtime.h"
#include <algorithm>
#include <stdio.h>
#include <time.h>

#include <hip/hip_runtime.h>

typedef float ftype;
typedef float2 vtype2;
typedef float4 vtype4;

#define BLOCKDIMX 128

void error_check(hipError_t cerr, int id) {
    if (cerr != hipSuccess) {
        fprintf(stderr, "Failed (%d) with error code %s\n", id, hipGetErrorString(cerr));
        exit(EXIT_FAILURE);
    }
}


// x [n_token * n_context * n_batch]
// wq [n_token * n_token]
// wk [n_token * n_token]
// wv [n_token * n_token]
// z [n_token * n_context * n_batch]
__global__ void attention_2x32(const int n,
                              const ftype __restrict__ *wq,
                              const ftype __restrict__ *wk,
                              const ftype __restrict__ *wv,
                              const ftype __restrict__ *x,
                              ftype __restrict__ *z) {
    int j = blockDim.x * blockIdx.x + threadIdx.x;
    int batch = j / 32;
    int l_batch = threadIdx.x / 32;
    int bld = l_batch*64;
    int i = threadIdx.x % warpSize;

    ftype __shared__ k_s[(BLOCKDIMX/32)*32*2], v_s[(BLOCKDIMX/32)*32*2];

    if (j < n) {
        // -> Load x into a float2
        // -> Load wq into a float4
        // -> Load wk into a float4
        // -> Load wv into a float4

        // Calculate key
        // -> replace the use of x and wk with the vector versions
        k_s[i + 0*32 + bld] =
        k_s[i + 1*32 + bld] =

        // Calculate value
        // -> replace the use of x and wv with the vector versions
        v_s[i + 0*32 + bld] =
        v_s[i + 1*32 + bld] =

        // Calculate query
        ftype q[2];
        // -> replace the use of x and wq with the vector versions
        q[0] =
        q[1] =

        __syncthreads();

        // Softmax
        ftype qkt[32], denom = 0.;
        for (int ii=0; ii<32; ++ii) {
            qkt[ii] = expf(q[0]*k_s[ii + 0*32 + bld] + q[1]*k_s[ii + 1*32 + bld]);
            denom += qkt[ii];
        }
        ftype rcp_denom = 1. / denom; // Why did I do this?? What is the benefit

        // Apply denom and value
        for (int iii=0; iii<2; ++iii) {
            ftype v = 0.;
            for (int ii=0; ii<32; ++ii) {
                v += v_s[ii + iii*32 + bld] * qkt[ii];
            }
            z[iii + 2*i + 64*batch] = v * rcp_denom;
        }
    }
}


int main(void) {
    hipError_t cerr;

    // Some constants and sizes
    const int nbatch = 1024;
    const int ncontext = 32;
    const int ntoken = 2;
    const int n = nbatch * ncontext * ntoken;
    size_t size_w = ntoken * ntoken * sizeof(ftype);
    size_t size_x = n * sizeof(ftype);
    size_t size_z = n * sizeof(ftype);

    // Allocate the host memory
    ftype *wq_h, *wk_h, *wv_h, *x_h, *z_h, *z_t;

    wq_h = (ftype *)malloc(size_w);
    wk_h = (ftype *)malloc(size_w);
    wv_h = (ftype *)malloc(size_w);
    x_h = (ftype *)malloc(size_x);
    z_h = (ftype *)malloc(size_z);
    z_t = (ftype *)malloc(size_z);

    // Initialise the weights on the host
    for (int i = 0; i < ntoken; ++i) {
        for (int ii = 0; ii < ntoken; ++ii) {
            wq_h[ii + ntoken*i] = 2*(rand() / (ftype)RAND_MAX) - 1;
            wk_h[ii + ntoken*i] = 2*(rand() / (ftype)RAND_MAX) - 1;
            wv_h[ii + ntoken*i] = 2*(rand() / (ftype)RAND_MAX) - 1;
        }
    }

    // Initialise the data
    for (int i = 0; i < n; ++i)
            x_h[i] = 2*(rand() / (ftype)RAND_MAX) - 1;

    // Allocate the device memory
    ftype *wq_d, *wk_d, *wv_d, *x_d, *z_d;
    cerr = hipMalloc((void **)&wq_d, size_w);
    error_check(cerr, 0);

    cerr = hipMalloc((void **)&wk_d, size_w);
    error_check(cerr, 1);

    cerr = hipMalloc((void **)&wv_d, size_w);
    error_check(cerr, 2);

    cerr = hipMalloc((void **)&x_d, size_x);
    error_check(cerr, 3);

    cerr = hipMalloc((void **)&z_d, size_z);
    error_check(cerr, 4);

    // Copy the weights, and data to the host
    cerr = hipMemcpy(wq_d, wq_h, size_w, hipMemcpyHostToDevice);
    error_check(cerr, 5);
    cerr = hipMemcpy(wk_d, wk_h, size_w, hipMemcpyHostToDevice);
    error_check(cerr, 6);
    cerr = hipMemcpy(wv_d, wv_h, size_w, hipMemcpyHostToDevice);
    error_check(cerr, 7);
    cerr = hipMemcpy(x_d, x_h, size_x, hipMemcpyHostToDevice);
    error_check(cerr, 8);
    cerr = hipMemcpy(z_d, z_h, size_z, hipMemcpyHostToDevice);
    error_check(cerr, 9);

    // Calcualte the thread block and grid sizes
    int threads_per_block = BLOCKDIMX;
    int batch_per_block = (BLOCKDIMX  + 31) / 32;
    int blocks_per_grid = (nbatch + batch_per_block - 1) / batch_per_block;
    printf("Threads per block: %d\nBlocks per grid: %d\n",
           threads_per_block,
           blocks_per_grid);

    // Launch the kernel
    attention_2x32<<<blocks_per_grid, threads_per_block>>>(n, wq_d, wk_d, wv_d, x_d, z_d);

    // Copy result to host
    cerr = hipMemcpy(z_h, z_d, size_z, hipMemcpyDeviceToHost);
    error_check(cerr, 10);

    // Free device memory
    cerr = hipFree(wq_d);
    error_check(cerr, 11);

    cerr = hipFree(wk_d);
    error_check(cerr, 12);

    cerr = hipFree(wv_d);
    error_check(cerr, 13);

    cerr = hipFree(x_d);
    error_check(cerr, 14);

    cerr = hipFree(z_d);
    error_check(cerr, 15);

    // Free host memory
    free(wq_h);
    free(wk_h);
    free(wv_h);
    free(x_h);
    free(z_h);
    free(z_t);

    return 0;
}
