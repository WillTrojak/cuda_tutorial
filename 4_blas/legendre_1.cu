#include <stdio.h>

#include <hipblas.h>
#include <hip/hip_runtime.h>


typedef float ftype;


void bstat_check(hipblasStatus_t stat, int id) {
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "Failed (%d) with error code %d\n", id, stat);
        exit(EXIT_FAILURE);
    }
}


void error_check(hipError_t cerr, int id) {
    if (cerr != hipSuccess) {
        fprintf(stderr, "Failed (%d) with error code %s\n", id, hipGetErrorString(cerr));
        exit(EXIT_FAILURE);
    }
}


void legendre(const int order, const float x, float * __restrict__ p, const int ldp) {
    if (order < 0) {
        fprintf(stderr, "legendre order should be zero or positive");
        exit(EXIT_FAILURE);
    }

    if (order >= 0)
        p[0] = 1.;

    if (order >= 1)
        p[ldp] = x;

    if (order > 1)
        for (int i = 2; i <= order; ++i)
            p[i*ldp] = ((2.*i - 1)*x*p[(i-1)*ldp] - (i - 1.)*p[(i-2)*ldp]) / i;
}


void projection_mat(const int n, const int order, const float *x,
                    float * __restrict__ mat) {
    for (int i = 0; i < n ; ++i){
        legendre(order, x[i], mat + i, n);
    }
}


int main(void) {
    hipError_t cerr;
    hipblasStatus_t bstat;

    int order = 2;
    int npnts = order + 1;
    int neles = 3;
    size_t size_proj = (order + 1) * npnts * sizeof(float);
    size_t size_pnts = npnts * sizeof(float);
    int nmode = neles * npnts;
    size_t size_mode = nmode * sizeof(float);

    // Allocate the host memory
    //   proj[npnts x order+1]
    //   mode[order+1 x neles]
    //   tran[npnts x neles]
    float *proj_h, *pnts_h, *tran_h;
    float *mode1_h, *mode2_h, *mode3_h;
    proj_h = (float *)malloc(size_proj);
    pnts_h = (float *)malloc(size_pnts);

    tran_h = (float *)malloc(3*size_mode);
    mode1_h = (float *)malloc(size_mode);
    mode2_h = (float *)malloc(size_mode);
    mode3_h = (float *)malloc(size_mode);

    // Initialise point vector
    for (int i = 0; i < npnts; ++i)
        pnts_h[i] = ((float)i) / ((float)npnts);

    // Initialise the mode values
    for (int i = 0; i < neles; ++i)
        for (int ii = 0; ii < npnts; ++ii) {
            mode1_h[i*npnts + ii] = rand() / (float)RAND_MAX;
            mode2_h[i*npnts + ii] = rand() / (float)RAND_MAX;
            mode3_h[i*npnts + ii] = rand() / (float)RAND_MAX;
        }

    // Initialise the projection matrix
    projection_mat(npnts, order, pnts_h, proj_h);

    // Allocate the device memory
    float *proj_d, *pnts_d, *tran_d;
    float *mode1_d, *mode2_d, *mode3_d;

    cerr = hipMalloc((void **)&tran_d, 3*size_mode);
    error_check(cerr, 0);

    cerr = hipMalloc((void **)&mode1_d, size_mode);
    error_check(cerr, 1);
    cerr = hipMalloc((void **)&mode2_d, size_mode);
    error_check(cerr, 2);
    cerr = hipMalloc((void **)&mode3_d, size_mode);
    error_check(cerr, 3);

    cerr = hipMalloc((void **)&proj_d, size_proj);
    error_check(cerr, 4);

    cerr = hipMalloc((void **)&pnts_d, size_pnts);
    error_check(cerr, 5);

    // Copy the matrices to the device
    cerr = hipMemcpy(mode1_d, mode1_h, size_mode, hipMemcpyHostToDevice);
    error_check(cerr, 6);
    cerr = hipMemcpy(mode2_d, mode2_h, size_mode, hipMemcpyHostToDevice);
    error_check(cerr, 7);
    cerr = hipMemcpy(mode3_d, mode3_h, size_mode, hipMemcpyHostToDevice);
    error_check(cerr, 8);
    cerr = hipMemcpy(pnts_d, pnts_h, size_pnts, hipMemcpyHostToDevice);
    error_check(cerr, 9);
    cerr = hipMemcpy(proj_d, proj_h, size_proj, hipMemcpyHostToDevice);
    error_check(cerr, 10);

    // Create a stream
    hipStream_t stream1, stream2, stream3;
    // -> Create 3 streams

    // Create a cublas handle and set the stream
    hipblasHandle_t handle1, handle2, handle3;
    // -> Create 3 cublas handle
    // -> Set the streams of the 3 handles

    // Set the alpha and beta value for the gemm
    float alpha = 1., beta = 0.;

    // Call the gemm function
    // C = alpha * A * B + beta * C
    // -> Call gemm 3 times but for each mode and with seperate hnadles.
    //    make sure you write the output to the corresponding part of tran

    // -> As we have 3 streams in flight call a syncronise

    // Copy result to host
    cerr = hipMemcpy(tran_h, tran_d, 3*size_mode, hipMemcpyDeviceToHost);
    error_check(cerr, 22);

    // Destroy the handle
    bstat = hipblasDestroy(handle1);
    bstat = hipblasDestroy(handle2);
    bstat = hipblasDestroy(handle3);

    // Free device memory
    cerr = hipFree(proj_d);
    error_check(cerr, 23);

    cerr = hipFree(pnts_d);
    error_check(cerr, 24);

    cerr = hipFree(mode1_d);
    error_check(cerr, 25);
    cerr = hipFree(mode2_d);
    error_check(cerr, 26);
    cerr = hipFree(mode3_d);
    error_check(cerr, 27);

    cerr = hipFree(tran_d);
    error_check(cerr, 28);

    // Free host memory
    free(pnts_h);
    free(proj_h);
    free(mode1_h);
    free(mode2_h);
    free(mode3_h);
    free(tran_h);

    return 0;
}
