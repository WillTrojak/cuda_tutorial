#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>

typedef double ftype;

// -> Update proto-types
ftype tricomi_approx(int n, int k);
ftype olver_approx(int n, int k);
ftype bessel0_root(const int k);


void error_check(hipError_t cerr, int id) {
    if (cerr != hipSuccess) {
        fprintf(stderr, "Failed (%d) with error code %s\n", id, hipGetErrorString(cerr));
        exit(EXIT_FAILURE);
    }
}

__global__ void invert_theta(int n, const ftype *theta, ftype *x) {
    int k = blockDim.x*blockIdx.x + threadIdx.x;
    int n2 = floor(n*0.5);

    if (k < n2) {
        ftype cos_theta = cos(theta[k]);
        x[k] = cos_theta;
        x[n - k - 1] = -cos_theta;
    }
}

__global__ void calculate_theta(int n, ftype *theta) {
    int k = blockDim.x*blockIdx.x + threadIdx.x;
    int n2 = floor(n*0.5);

    if (k < n2) {
        ftype x_k = tricomi_approx(n, k+1);
        if(fabs(x_k) > 0.5) x_k = olver_approx(n, k+1);

        theta[k] = acos(x_k);
    }

}

// -> Update proto-types
ftype tricomi_approx(int n, int k) { // best for |x| < 0.5
    const ftype pi = 3.141592653589793;

    ftype phi = (k - 0.25)*pi / (n + 0.5);

    ftype x_k = 39 - 28/(pow(sin(phi), 2));
    x_k = -x_k / (384*pow(n, 4));
    x_k -= (n - 1.) / (8*pow(n, 3));
    x_k += 1;
    x_k = x_k * cos(phi);

    return x_k;
}

// -> Update proto-types
ftype olver_approx(int n, int k) { // best for 0.5 <= |x| <=1
    ftype nph = n + 0.5;
    ftype psi = bessel0_root(k) / nph;

    ftype x_k = psi / tan(psi) - 1;
    x_k = x_k / (8 * psi * nph * nph);
    x_k += psi;
    x_k = cos(x_k);

    return x_k;
}

// -> Implement approximate bessel_0 root function
// https://doi.org/10.1016/0021-9991(81)90253-9
// Hints:
// pi = 3.141592653589793
// beta = (k - 0.25) * pi;
ftype bessel0_root(int k) {

    return
}

int main(void) {
    hipError_t cerr;

    int n = 11;
    size_t size = n * sizeof(ftype);

    // Allocate the host memory
    ftype *theta_h = (ftype *)malloc(size);
    ftype *x_h = (ftype *)malloc(size);

    // Allocate the device memory
    ftype *theta_d, *x_d;
    cerr = hipMalloc((void **)&theta_d, size);
    error_check(cerr, 0);
    cerr = hipMalloc((void **)&x_d, size);
    error_check(cerr, 1);

    // Create the stream
    // -> Declare and create a stream
    error_check(cerr, 2);

    // Calculate the thread size
    int threads_per_block = 256;
    int nh = floor(n*0.5);
    int blocks_per_grid = (nh + threads_per_block - 1) / threads_per_block;
    printf("Threads per block: %d\nBlocks per grid: %d\n",
           threads_per_block,
           blocks_per_grid);

    // Launch the CUDA Kernels
    // -> call calculate_theta
    // -> call invert_theta

    // Copy result to host
    cerr = hipMemcpy(theta_h, theta_d, size, hipMemcpyDeviceToHost);
    error_check(cerr, 3);
    cerr = hipMemcpy(x_h, x_d, size, hipMemcpyDeviceToHost);
    error_check(cerr, 4);

    // Print the result
    if (n < 20) {
        for (int i = 0; i < n; ++i) {
            printf("x_h[%d]: %f\n", i, x_h[i]);
        }
    }

    // Free device memory
    cerr = hipFree(theta_d);
    error_check(cerr, 5);

    cerr = hipFree(x_d);
    error_check(cerr, 6);

    // Free host memory
    free(theta_h);
    free(x_h);

    return 0;
}
